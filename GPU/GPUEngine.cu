#include "hip/hip_runtime.h"
/*
 * This file is part of the BTCCollider distribution (https://github.com/JeanLucPons/BTCCollider).
 * Copyright (c) 2020 Jean Luc PONS.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef WIN64
#include <unistd.h>
#include <stdio.h>
#endif

#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdint.h>
#include "../hash/sha256.h"
#include "../hash/ripemd160.h"
#include "../Timer.h"

#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUCompute.h"

// ---------------------------------------------------------------------------------------

__global__ void comp_hash(uint64_t *keys, uint64_t *hashes, uint32_t maxFound, uint32_t *found, uint64_t dpMask, uint16_t colMask, uint16_t nbFull, bool extraPoints) {

  int xPtr = (blockIdx.x*blockDim.x*GPU_GRP_SIZE) * 6;
  ComputeHash(keys, hashes + xPtr, maxFound, found, dpMask, colMask, nbFull, extraPoints);

}

__global__ void comp_hash_p2sh(uint64_t *keys, uint64_t *hashes, uint32_t maxFound, uint32_t *found, uint64_t dpMask, uint16_t colMask, uint16_t nbFull, bool extraPoints) {

  int xPtr = (blockIdx.x*blockDim.x*GPU_GRP_SIZE) * 6;
  ComputeHashP2SH(keys, hashes + xPtr, maxFound, found, dpMask, colMask, nbFull, extraPoints);

}

// ---------------------------------------------------------------------------------------

using namespace std;

int _ConvertSMVer2Cores(int major, int minor) {

  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
      {0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {-1, -1} };

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  return 0;

}

GPUEngine::GPUEngine(int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound) {

  // Initialise CUDA
  this->nbThreadPerGroup = nbThreadPerGroup;
  initialised = false;
  hipError_t err;

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("GPUEngine: CudaGetDeviceCount %s\n", hipGetErrorString(error_id));
    return;
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0) {
    printf("GPUEngine: There are no available device(s) that support CUDA\n");
    return;
  }

  err = hipSetDevice(gpuId);
  if (err != hipSuccess) {
    printf("GPUEngine: %s\n", hipGetErrorString(err));
    return;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, gpuId);

  this->nbThread = nbThreadGroup * nbThreadPerGroup;
  this->maxFound = maxFound;
  this->outputSize = (maxFound*ITEM_SIZE + 4);

  char tmp[512];
  sprintf(tmp,"GPU #%d %s (%dx%d cores) Grid(%dx%d)",
  gpuId,deviceProp.name,deviceProp.multiProcessorCount,
  _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
                      nbThread / nbThreadPerGroup,
                      nbThreadPerGroup);
  deviceName = std::string(tmp);

  // Prefer L1 (We do not use __shared__ at all)
  err = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  if (err != hipSuccess) {
    printf("GPUEngine: %s\n", hipGetErrorString(err));
    return;
  }

  /*
  size_t stackSize = 49152;
  err = hipDeviceSetLimit(hipLimitStackSize, stackSize);
  if (err != hipSuccess) {
    printf("GPUEngine: %s\n", hipGetErrorString(err));
    return;
  }

  size_t heapSize = ;
  err = hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize);
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    exit(0);
  }

  size_t size;
  hipDeviceGetLimit(&size, hipLimitStackSize);
  printf("Stack Size %lld\n", size);
  hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
  printf("Heap Size %lld\n", size);
  */

  // Allocate memory
  inputKey = NULL;
  inputKeyPinned = NULL;
  outputHash = NULL;
  outputHashPinned = NULL;
  inputHash = NULL;
  inputHashPinned = NULL;

  // Input keys (see BTCCollider.cpp)
  keySize = 10 * _64K * 32 * 2;
  err = hipMalloc((void **)&inputKey, keySize);
  if (err != hipSuccess) {
    printf("GPUEngine: Allocate input memory: %s\n", hipGetErrorString(err));
    return;
  }
  err = hipHostAlloc(&inputKeyPinned, keySize, hipHostMallocWriteCombined | hipHostMallocMapped);
  if (err != hipSuccess) {
    printf("GPUEngine: Allocate input pinned memory: %s\n", hipGetErrorString(err));
    return;
  }
  // OutputHash
  err = hipMalloc((void **)&outputHash, outputSize);
  if (err != hipSuccess) {
    printf("GPUEngine: Allocate output memory: %s\n", hipGetErrorString(err));
    return;
  }
  err = hipHostAlloc(&outputHashPinned, outputSize, hipHostMallocMapped);
  if (err != hipSuccess) {
    printf("GPUEngine: Allocate output pinned memory: %s\n", hipGetErrorString(err));
    return;
  }
  // InputHash (hash160 sotred on 3*64bit)
  inputHashSize = GPU_GRP_SIZE * nbThread * 48;
  err = hipMalloc((void **)&inputHash, inputHashSize);
  if (err != hipSuccess) {
    printf("GPUEngine: Allocate output memory: %s\n", hipGetErrorString(err));
    return;
  }
  err = hipHostAlloc(&inputHashPinned, inputHashSize, hipHostMallocWriteCombined | hipHostMallocMapped);
  if (err != hipSuccess) {
    printf("GPUEngine: Allocate output pinned memory: %s\n", hipGetErrorString(err));
    return;
  }

  searchType = P2PKH;
  initialised = true;
  useExtraPoints = false;


}

GPUEngine::~GPUEngine() {

  if(inputKey) hipFree(inputKey);
  if(inputHash) hipFree(inputHash);
  if(outputHash) hipFree(outputHash);
  if(inputKeyPinned) hipHostFree(inputKeyPinned);
  if(inputHashPinned) hipHostFree(inputHashPinned);
  if(outputHashPinned) hipHostFree(outputHashPinned);

}


int GPUEngine::GetMemory() {
  return keySize + inputHashSize + outputSize;
}


int GPUEngine::GetGroupSize() {
  return GPU_GRP_SIZE;
}

bool GPUEngine::GetGridSize(int gpuId, int *x, int *y) {

  if ( *x <= 0 || *y <= 0 ) {

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
      printf("GPUEngine: CudaGetDeviceCount %s\n", hipGetErrorString(error_id));
      return false;
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0) {
      printf("GPUEngine: There are no available device(s) that support CUDA\n");
      return false;
    }

    if (gpuId >= deviceCount) {
      printf("GPUEngine::GetGridSize() Invalid gpuId\n");
      return false;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, gpuId);

    if(*x<=0) *x = 2 * deviceProp.multiProcessorCount;
    if(*y<=0) *y = 2 * _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    if(*y<=0) *y = 128;

  }

  return true;

}

void *GPUEngine::AllocatePinnedMemory(size_t size) {

  void *buff;

  hipError_t err = hipHostAlloc(&buff, size, hipHostMallocPortable);
  if (err != hipSuccess) {
    printf("GPUEngine: AllocatePinnedMemory: %s\n", hipGetErrorString(err));
    return NULL;
  }

  return buff;

}

void GPUEngine::FreePinnedMemory(void *buff) {
  hipHostFree(buff);
}

void GPUEngine::PrintCudaInfo() {

  hipError_t err;

  const char *sComputeMode[] =
  {
    "Multiple host threads",
    "Only one host thread",
    "No host thread",
    "Multiple process threads",
    "Unknown",
     NULL
  };

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("GPUEngine: CudaGetDeviceCount %s\n", hipGetErrorString(error_id));
    return;
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0) {
    printf("GPUEngine: There are no available device(s) that support CUDA\n");
    return;
  }

  for(int i=0;i<deviceCount;i++) {

    err = hipSetDevice(i);
    if (err != hipSuccess) {
      printf("GPUEngine: hipSetDevice(%d) %s\n", i, hipGetErrorString(err));
      return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, i);
    printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
      i,deviceProp.name,deviceProp.multiProcessorCount,
      _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
      deviceProp.major, deviceProp.minor,(double)deviceProp.totalGlobalMem/1048576.0,
      sComputeMode[deviceProp.computeMode]);

  }

}

void GPUEngine::SetExtraPoint(bool extraPoint) {
  useExtraPoints = extraPoint;
}

int GPUEngine::GetNbThread() {
  return nbThread;
}

void GPUEngine::SetSearchType(int searchType) {
  this->searchType = searchType;
}

bool GPUEngine::GetHashes(uint64_t *sHash, uint64_t *cHash) {

  // Retrieve hash from device memory
  hipMemcpy(inputHashPinned, inputHash, inputHashSize, hipMemcpyDeviceToHost);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("GPUEngine: GetHashes: %s\n", hipGetErrorString(err));
    return false;
  }

  int gSize = 6 * GPU_GRP_SIZE;
  int strideSize = nbThreadPerGroup * 6;
  int nbBlock = nbThread / nbThreadPerGroup;
  int blockSize = nbThreadPerGroup * gSize;

  for (int b = 0; b < nbBlock; b++) {
    for (int g = 0; g < GPU_GRP_SIZE; g++) {
      for (int t = 0; t < nbThreadPerGroup; t++) {
        // Current hash
        cHash[0] = inputHashPinned[b * blockSize + g * strideSize + t + 0 * nbThreadPerGroup];
        cHash[1] = inputHashPinned[b * blockSize + g * strideSize + t + 1 * nbThreadPerGroup];
        cHash[2] = inputHashPinned[b * blockSize + g * strideSize + t + 2 * nbThreadPerGroup];
        // Start hash
        sHash[0] = inputHashPinned[b * blockSize + g * strideSize + t + 3 * nbThreadPerGroup];
        sHash[1] = inputHashPinned[b * blockSize + g * strideSize + t + 4 * nbThreadPerGroup];
        sHash[2] = inputHashPinned[b * blockSize + g * strideSize + t + 5 * nbThreadPerGroup];
        cHash += 3;
        sHash += 3;
      }
    }
  }

  return true;

}

bool GPUEngine::SetStartingHashes(uint64_t *sHash, uint64_t *cHash) {

  lostWarning = false;

  // Sets the starting hash of each thread
  int gSize = 6 * GPU_GRP_SIZE;
  int strideSize = nbThreadPerGroup * 6;
  int nbBlock = nbThread / nbThreadPerGroup;
  int blockSize = nbThreadPerGroup * gSize;

  for (int b = 0; b < nbBlock; b++) {
    for (int g = 0; g < GPU_GRP_SIZE; g++) {
      for (int t = 0; t < nbThreadPerGroup; t++) {
        // Current hash
        inputHashPinned[b * blockSize + g * strideSize + t + 0* nbThreadPerGroup] = cHash[0];
        inputHashPinned[b * blockSize + g * strideSize + t + 1* nbThreadPerGroup] = cHash[1];
        inputHashPinned[b * blockSize + g * strideSize + t + 2* nbThreadPerGroup] = cHash[2];
        // Start hash
        inputHashPinned[b * blockSize + g * strideSize + t + 3 * nbThreadPerGroup] = sHash[0];
        inputHashPinned[b * blockSize + g * strideSize + t + 4 * nbThreadPerGroup] = sHash[1];
        inputHashPinned[b * blockSize + g * strideSize + t + 5 * nbThreadPerGroup] = sHash[2];
        cHash += 3;
        sHash += 3;
      }
    }
  }

  // Fill device memory
  hipMemcpy(inputHash, inputHashPinned, inputHashSize, hipMemcpyHostToDevice);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("GPUEngine: SetStartHashes: %s\n", hipGetErrorString(err));
  }

  return callKernel();

}

bool GPUEngine::callKernel() {

  // Reset nbFound
  hipMemset(outputHash,0,4);

  // Call the kernel (Perform STEP_SIZE keys per thread)
  if (searchType == P2SH) {

    comp_hash_p2sh << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
        (inputKey, inputHash, maxFound, outputHash, dpMask, colMask, nbFull, useExtraPoints);

  } else {

    // P2PKH or BECH32
    comp_hash << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
      (inputKey, inputHash, maxFound, outputHash, dpMask, colMask, nbFull, useExtraPoints);
    
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("GPUEngine: Kernel: %s\n", hipGetErrorString(err));
    return false;
  }
  return true;

}

void GPUEngine::SetMasks(uint16_t colMask, uint64_t dpMask, uint16_t nbFull) {
  this->colMask = colMask;
  this->dpMask = dpMask;
  this->nbFull = nbFull;
}

#define PX(i,j) p[(i)*(65536*2) + 2*(j)]
#define PY(i,j) p[(i)*(65536*2) + 2*(j)+1]

void GPUEngine::SetKeys(Int *p) {

  // Sets the base keys for mapping

  for (int i = 0; i < 10; i++) {
    for (int j = 0; j < 65536; j++) {

      inputKeyPinned[8 * ((i*_64K) + j) + 0] = PX(i, j).bits64[0];
      inputKeyPinned[8 * ((i*_64K) + j) + 1] = PX(i, j).bits64[1];
      inputKeyPinned[8 * ((i*_64K) + j) + 2] = PX(i, j).bits64[2];
      inputKeyPinned[8 * ((i*_64K) + j) + 3] = PX(i, j).bits64[3];

      inputKeyPinned[8 * ((i*_64K) + j) + 4] = PY(i, j).bits64[0];
      inputKeyPinned[8 * ((i*_64K) + j) + 5] = PY(i, j).bits64[1];
      inputKeyPinned[8 * ((i*_64K) + j) + 6] = PY(i, j).bits64[2];
      inputKeyPinned[8 * ((i*_64K) + j) + 7] = PY(i, j).bits64[3];

    }
  }

  // Fill device memory
  hipMemcpy(inputKey, inputKeyPinned, keySize, hipMemcpyHostToDevice);

  // We do not need the key pinned memory anymore
  hipHostFree(inputKeyPinned);
  inputKeyPinned = NULL;

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("GPUEngine: SetKeys: %s\n", hipGetErrorString(err));
  }

}

bool GPUEngine::Launch(std::vector<ITEM> &hashFound,bool spinWait) {


  hashFound.clear();

  // Get the result

  if(spinWait) {

    hipMemcpy(outputHashPinned, outputHash, outputSize, hipMemcpyDeviceToHost);

  } else {

    // Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
    hipEvent_t evt;
    hipEventCreate(&evt);
    hipMemcpyAsync(outputHashPinned, outputHash, 4, hipMemcpyDeviceToHost, 0);
    hipEventRecord(evt, 0);
    while (hipEventQuery(evt) == hipErrorNotReady) {
      // Sleep 1 ms to free the CPU
      Timer::SleepMillis(1);
    }
    hipEventDestroy(evt);

  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("GPUEngine: Launch: %s\n", hipGetErrorString(err));
    return false;
  }

  // Look for prefix found
  uint32_t nbFound = outputHashPinned[0];
  if (nbFound > maxFound) {
    // prefix has been lost
    if (!lostWarning) {
      printf("\nWarning, %d items lost\nHint: Search with less threads (-g)\n", (nbFound - maxFound));
      lostWarning = true;
    }
    nbFound = maxFound;
  }
  
  // When can perform a standard copy, the kernel is eneded
  hipMemcpy(outputHashPinned, outputHash, nbFound*ITEM_SIZE + 4 , hipMemcpyDeviceToHost);
  
  for (uint32_t i = 0; i < nbFound; i++) {
    uint32_t *itemPtr = outputHashPinned + (i*ITEM_SIZE32 + 1);
    ITEM it;
    it.h1 = ((uint8_t *)(itemPtr));
    it.h2 = ((uint8_t *)(itemPtr))+20;
    hashFound.push_back(it);
  }

  return callKernel();

}
